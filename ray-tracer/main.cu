#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <time.h>

#include "ray.cuh"
#include "scene.cuh"
#include "material.cuh"
#include "shapes.cuh"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}
/*
__global__ void render(float* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}
*/
__global__ void create_world(SceneObject** sceneObjects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(sceneObjects) = new Sphere(vec3(600, 400, 400), 200);
    }
}

__global__ void free_world(SceneObject** sceneObjects) {
    for (int i = 0; i < 1; i++) {
        //delete ((sphere*)d_list[i])->mat_ptr;
    }
    delete* sceneObjects;
}

__device__ vec3 color(const ray& r, SceneObject** sceneObjects) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 1; i++) {
        isect is;
        //go through all scene objects
        
        for (int j = 0; j < 1; j++) {
            if (sceneObjects[j]->intersects(cur_ray, 0.001f, FLT_MAX, is)) {
                cur_attenuation = vec3(0, 0, 0);
            }
        }
    }
    return cur_attenuation;
}

__global__ void render(vec3* fb, int max_x, int max_y, SceneObject** sceneObjects) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    vec3 col(0, 0, 0);
    vec3 cameraPos(600, 400, -400);
    vec3 screenPos(i, j, 0);
    ray r(cameraPos, screenPos - cameraPos);
    col = color(r, sceneObjects);
    fb[pixel_index] = col;
}


int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    //screen
    //lower left is (0, 0, 0), screen plane is x and y axis
    
    SceneObject** sceneObjects;
    int numObjects = 1;
    checkCudaErrors(hipMalloc((void**)&sceneObjects, numObjects * sizeof(SceneObject*)));
    create_world << <1, 1 >> > (sceneObjects);
    //sceneObjects[0]->test();

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    //render << <blocks, threads >> > (fb, nx, ny);
    render << <blocks, threads >> > (fb, nx, ny, sceneObjects);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image

    std::ofstream outfile("test.ppm");

    // Output FB as Image
    outfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }
    

    checkCudaErrors(hipFree(fb));
    free_world << <1, 1 >> > (sceneObjects);
}