#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <cfloat>
#include <ctime>

#include "check.cuh"
#include "ray.cuh"
#include "scene.cuh"
#include "light.cuh"
#include "material.cuh"
#include "shapes.cuh"
#include "parser.cuh"
#include "trimesh.cuh"

__global__ void create_world(SceneObject** sceneObjects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //material mat1(vec3(1.0, 1.0, 1.0), vec3(1.0, 1.0, 1.0));
        material* mat1 = new material(vec3(0.4, 0.6, 0.3), vec3(0.7, 1.0, 0.5), false);
        //material mat2(vec3(1.0, 0.35, 0.5), vec3(1.0, 0.35, 0.5));
        *(sceneObjects) = new Floor();
        *(sceneObjects + 1) = new Sphere(vec3(600, 400, 400), 200, mat1);
        *(sceneObjects + 2) = new Box(vec3(1000, 10, 400), vec3(1300, 310, 700), mat1);
    }
}

__global__ void free_world(SceneObject** sceneObjects) {
    for (int i = 0; i < 1; i++) {
        //delete ((sphere*)d_list[i])->mat_ptr;
    }
    delete* sceneObjects;
}

__device__ vec3 color(const ray& r, Scene* scene) {
    ray cur_ray = r;
    vec3 background = vec3(0.1, 0.65, 1.0);
    vec3 total = vec3(0.0, 0.0, 0.0);
    //TODO - add kr into material, it is hardcoded into the constructor rn
    vec3 kr_factor = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 2; i++) {
        isect is;    
        if (scene->intersects(cur_ray, 0.001f, FLT_MAX, is)) {
            total += kr_factor * is.mat_ptr->shade(scene, r, is);
            if (!is.mat_ptr->refl) {
                break;
            }
            kr_factor *= is.mat_ptr->kr;
            vec3 reflect_dir = cur_ray.direction() - 2 * dot(cur_ray.direction(), is.normal) * is.normal;
            cur_ray = ray(is.p, unit_vector(reflect_dir));
        } else {
            // TODO: get background from scene method
            total += kr_factor * background;
            break;
        }
    }
    return clamp(total);
}

__global__ void render(vec3* fb, int max_x, int max_y, Scene* scene) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    
    vec3 cameraPos(600, 400, -400);
    vec3 look = vec3(0, 0, 1);  // can change to whatever look direction
    vec3 u(1, 0, 0), v(0, 1, 0);
    //float x = (i + 0.5) / max_x - 0.5;  // normalized to [-0.5, 0.5]
    //float y = (j + 0.5) / max_y - 0.5;
    //ray r(cameraPos, unit_vector(look + (x * u) + (y * v)));
    
    float fov = 30;
    float aspectratio = max_x / float(max_y);
    // float M_PI = 3.141592653589793;
    float angle = tan(M_PI * 0.5 * fov / 180);
    float xx = (2 * ((i + 0.5) / float(max_x)) - 1) * angle * aspectratio;
    float yy = (1 - 2 * ((j + 0.5) / float(max_y))) * angle;
    vec3 dir = unit_vector(vec3(xx, -yy, 0.5));
    ray r(cameraPos, dir);
    
    vec3 col = color(r, scene);
    int pixel_index = j * max_x + i;
    fb[pixel_index] = col;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    //parse
    Scene* scene = parse();

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    //screen
    //lower left is (0, 0, 0), screen plane is x and y axis
    
    // int numObjects = 3;
    // checkCudaErrors(hipMalloc((void**)&sceneObjects, numObjects * sizeof(SceneObject*)));
    // create_world<<<1, 1>>>(sceneObjects);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
    dim3 threads(tx, ty);
    //render << <blocks, threads >> > (fb, nx, ny);
    render<<<blocks, threads>>>(fb, nx, ny, scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image

    std::ofstream outfile("test.ppm");

    // Output FB as Image
    outfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    checkCudaErrors(hipFree(fb));
    // free_world << <1, 1 >> > (sceneObjects);
}

