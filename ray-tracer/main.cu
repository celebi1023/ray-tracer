#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <cfloat>
#include <ctime>

#include "check.cuh"
#include "ray.cuh"
#include "scene.cuh"
#include "light.cuh"
#include "material.cuh"
#include "shapes.cuh"
#include "parser.cuh"
#include "trimesh.cuh"

__global__ void create_world(SceneObject** sceneObjects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //material mat1(vec3(1.0, 1.0, 1.0), vec3(1.0, 1.0, 1.0));
        //material* mat1 = new material(vec3(0.4, 0.6, 0.3), vec3(0.7, 1.0, 0.5), false);
        //material mat2(vec3(1.0, 0.35, 0.5), vec3(1.0, 0.35, 0.5));
        //*(sceneObjects) = new Floor();
        //*(sceneObjects + 1) = new Sphere(vec3(600, 400, 400), 200, mat1);
        //*(sceneObjects + 2) = new Box(vec3(1000, 10, 400), vec3(1300, 310, 700), mat1);
    }
}

__global__ void free_world(SceneObject** sceneObjects) {
    for (int i = 0; i < 1; i++) {
        //delete ((sphere*)d_list[i])->mat_ptr;
    }
    delete* sceneObjects;
}

__device__ vec3 color(const ray& r, Scene* scene) {
    ray cur_ray = r;
    vec3 background = vec3(0.1, 0.65, 1.0);
    vec3 total = vec3(0.0, 0.0, 0.0);
    //TODO - add kr into material, it is hardcoded into the constructor rn
    vec3 k_factor = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 2; i++) {
        isect is;    
        if (scene->intersects(cur_ray, RAY_EPSILON, FLT_MAX, is)) {
            total += k_factor * is.mat_ptr->shade(scene, r, is);
            if (is.mat_ptr->trans) {
                bool inside = dot(cur_ray.direction(), is.normal) > 0;
                float n_i = inside ? is.mat_ptr->index : 1.0001;
                float n_t = inside ? 1.0001 : is.mat_ptr->index;
                vec3 normal = inside ? -is.normal : is.normal;
                float n_ratio = n_i / n_t;
                float cos_i = dot(-cur_ray.direction(), normal);
                float cos2_t = 1 - n_ratio * n_ratio * (1 - cos_i * cos_i);

                vec3 refract_dir;
                if (cos2_t < 0) {
                    //total internal refraction
                    refract_dir = cur_ray.direction() - 2 * dot(cur_ray.direction(), normal) * normal;
                }
                else {
                    float cos_t = sqrt(cos2_t);
                    refract_dir = (n_ratio * cos_i - cos_t) * normal - n_ratio * -cur_ray.direction();
                }

                if (inside) {
                    vec3 kt = is.mat_ptr->kt;
                    kt = vec3(pow(kt.x, is.t), pow(kt.y, is.t), pow(kt.z, is.t));
                    k_factor *= kt;
                }

                cur_ray = ray(cur_ray.at(is.t - RAY_EPSILON), unit_vector(refract_dir));
            }
            else if (is.mat_ptr->refl) {
                k_factor *= is.mat_ptr->kr;
                vec3 reflect_dir = cur_ray.direction() - 2 * dot(cur_ray.direction(), is.normal) * is.normal;
                cur_ray = ray(cur_ray.at(is.t - RAY_EPSILON), unit_vector(reflect_dir));
            }
            else {
                break;
            }
        } else {
            // TODO: get background from scene method
            total += k_factor * background;
            break;
        }
    }
    return clamp(total);
}

__global__ void render(vec3* fb, int max_x, int max_y, Scene* scene) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    
    vec3 cameraPos(600, 400, -400);
    vec3 look = vec3(0, 0, 1);  // can change to whatever look direction
    vec3 u(1, 0, 0), v(0, 1, 0);
    //float x = (i + 0.5) / max_x - 0.5;  // normalized to [-0.5, 0.5]
    //float y = (j + 0.5) / max_y - 0.5;
    //ray r(cameraPos, unit_vector(look + (x * u) + (y * v)));
    
    float fov = 30;
    float aspectratio = max_x / float(max_y);
    // float M_PI = 3.141592653589793;
    float angle = tan(M_PI * 0.5 * fov / 180);
    float xx = (2 * ((i + 0.5) / float(max_x)) - 1) * angle * aspectratio;
    float yy = (1 - 2 * ((j + 0.5) / float(max_y))) * angle;
    vec3 dir = unit_vector(vec3(xx, -yy, 0.5));
    ray r(cameraPos, dir);
    
    vec3 col = color(r, scene);
    int pixel_index = j * max_x + i;
    fb[pixel_index] = col;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    //parse
    Scene* scene = parse();

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    //screen
    //lower left is (0, 0, 0), screen plane is x and y axis
    
    // int numObjects = 3;
    // checkCudaErrors(hipMalloc((void**)&sceneObjects, numObjects * sizeof(SceneObject*)));
    // create_world<<<1, 1>>>(sceneObjects);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks((nx + tx - 1) / tx, (ny + ty - 1) / ty);
    dim3 threads(tx, ty);
    //render << <blocks, threads >> > (fb, nx, ny);
    render<<<blocks, threads>>>(fb, nx, ny, scene);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image

    std::ofstream outfile("test.ppm");

    // Output FB as Image
    outfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    checkCudaErrors(hipFree(fb));
    // free_world << <1, 1 >> > (sceneObjects);
}

